#include "hip/hip_runtime.h"
#include "stdio.h"
#include<iostream>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
//Defining number of elements in Array
#define N	50000

//Defining Kernel function for vector addition
__global__ void gpuAdd(int *d_a, int *d_b, int *d_c) {
	//Getting Thread index of current kernel

	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	while (tid < N)
	{
		d_c[tid] = d_a[tid] + d_b[tid];
		tid += blockDim.x * gridDim.x;
	}

}

int main(void) {
	//Defining host arrays
	int *h_a, *h_b, *h_c;
	//Defining device pointers for stream 0
	int *d_a0, *d_b0, *d_c0;
	//Defining device pointers for stream 1
	int *d_a1, *d_b1, *d_c1;
	hipStream_t stream0, stream1;
	hipStreamCreate(&stream0);
	hipStreamCreate(&stream1);
	hipEvent_t e_start, e_stop;
	hipEventCreate(&e_start);
	hipEventCreate(&e_stop);
	hipEventRecord(e_start, 0);


	hipHostAlloc((void**)&h_a,N *2* sizeof(int),hipHostMallocDefault);
	hipHostAlloc((void**)&h_b, N *2* sizeof(int), hipHostMallocDefault);
	hipHostAlloc((void**)&h_c, N *2*sizeof(int), hipHostMallocDefault);


	// allocate the memory
	hipMalloc((void**)&d_a0, N * sizeof(int));
	hipMalloc((void**)&d_b0, N * sizeof(int));
	hipMalloc((void**)&d_c0, N * sizeof(int));
	hipMalloc((void**)&d_a1, N * sizeof(int));
	hipMalloc((void**)&d_b1, N * sizeof(int));
	hipMalloc((void**)&d_c1, N * sizeof(int));
	//Initializing Arrays
	for (int i = 0; i < N*2; i++) {
		h_a[i] = 2 * i*i;
		h_b[i] = i;
	}
	
		hipMemcpyAsync(d_a0, h_a , N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(d_a1, h_a+ N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		hipMemcpyAsync(d_b0, h_b , N * sizeof(int), hipMemcpyHostToDevice, stream0);
		hipMemcpyAsync(d_b1, h_b + N, N * sizeof(int), hipMemcpyHostToDevice, stream1);
		//Calling kernels passing device pointers as parameters
		gpuAdd << <512, 512, 0, stream0 >> > (d_a0, d_b0, d_c0);
		gpuAdd << <512, 512, 0, stream1 >> > (d_a1, d_b1, d_c1);
		//Copy result back to host memory from device memory
		hipMemcpyAsync(h_c , d_c0, N * sizeof(int), hipMemcpyDeviceToHost, stream0);
		hipMemcpyAsync(h_c + N, d_c1, N * sizeof(int), hipMemcpyDeviceToHost, stream1);

	hipDeviceSynchronize();
	hipStreamSynchronize(stream0);
	hipStreamSynchronize(stream1);
	hipEventRecord(e_stop, 0);
	hipEventSynchronize(e_stop);
	float elapsedTime;
	hipEventElapsedTime(&elapsedTime, e_start, e_stop);
	printf("Time to add %d numbers: %3.1f ms\n",2* N, elapsedTime);

	int Correct = 1;
	printf("Vector addition on GPU \n");
	//Printing result on console
	for (int i = 0; i < 2*N; i++) {
		if ((h_a[i] + h_b[i] != h_c[i]))
		{
			Correct = 0;
		}

	}
	if (Correct == 1)
	{
		printf("GPU has computed Sum Correctly\n");
	}
	else
	{
		printf("There is an Error in GPU Computation\n");
	}
	//Free up memory
	hipFree(d_a0);
	hipFree(d_b0);
	hipFree(d_c0);
	hipFree(d_a1);
	hipFree(d_b1);
	hipFree(d_c1);
	hipHostFree(h_a);
	hipHostFree(h_b);
	hipHostFree(h_c);
	return 0;
}
